#include "hip/hip_runtime.h"
/*!
 * Copyright 2018 H2O.ai, Inc.
 * License   Apache License Version 2.0 (see LICENSE for details)
 */
#include <algorithm>
#include <cmath>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/transform_reduce.h>
#include <thrust/fill.h>
#include <thrust/execution_policy.h>
#include "batching_gpu.cuh"
#include "../utils/hip/hip_runtime.h"
#include "../../base/ffm/trainer.h"
#include "../../common/timer.h"
#include "model_gpu.cuh"

#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600
#else
__device__ double atomicAdd(double* address, double val)
{
  unsigned long long int* address_as_ull =
      (unsigned long long int*)address;
  unsigned long long int old = *address_as_ull, assumed;
  do {
    assumed = old;
    old = atomicCAS(address_as_ull, assumed,
                    __double_as_longlong(val +
                        __longlong_as_double(assumed)));
  } while (assumed != old);
  return __longlong_as_double(old);
}
#endif

#define MAX_BLOCK_THREADS 128

namespace ffm {

template<typename T>
Trainer<T>::Trainer(Params &params) : params(params), trainDataBatcher(params.nGpus) {
  CUDA_CHECK(hipSetDeviceFlags(hipDeviceMapHost));
  CUDA_CHECK(hipSetDeviceFlags(hipDeviceScheduleSpin));

#if __CUDA_ARCH__ > 500
  CUDA_CHECK(hipDeviceSetLimit(cudaLimitDevRuntimeSyncDepth, 0));
  CUDA_CHECK(hipDeviceSetLimit(cudaLimitDevRuntimePendingLaunchCount, 0));
#endif // CUDA 5.0
  CUDA_CHECK(hipDeviceSetLimit(hipLimitStackSize, 0));
  CUDA_CHECK(hipDeviceSetLimit(hipLimitPrintfFifoSize, 0));
  CUDA_CHECK(hipDeviceSetCacheConfig(hipFuncCachePreferL1));

  this->model = new ModelGPU<T>(params);
}

template<typename T>
Trainer<T>::Trainer(const T* weights, Params &params) : params(params), trainDataBatcher(params.nGpus) {
  CUDA_CHECK(hipSetDeviceFlags(hipDeviceMapHost));
  CUDA_CHECK(hipSetDeviceFlags(hipDeviceScheduleSpin));

#if __CUDA_ARCH__ > 500
  CUDA_CHECK(hipDeviceSetLimit(cudaLimitDevRuntimeSyncDepth, 0));
  CUDA_CHECK(hipDeviceSetLimit(cudaLimitDevRuntimePendingLaunchCount, 0));
#endif // CUDA 5.0
  CUDA_CHECK(hipDeviceSetLimit(hipLimitStackSize, 0));
  CUDA_CHECK(hipDeviceSetLimit(hipLimitPrintfFifoSize, 0));
  CUDA_CHECK(hipDeviceSetCacheConfig(hipFuncCachePreferL1));

  this->model = new ModelGPU<T>(params, weights);
}

template<typename T>
void Trainer<T>::setDataset(const Dataset<T> &dataset) {
  DatasetBatcherGPU<T> *batcher = new DatasetBatcherGPU<T>(dataset, params);
  trainDataBatcher[0] = batcher;
}

template<typename T>
Trainer<T>::~Trainer() {
  delete trainDataBatcher[0];
  delete model;
  CUDA_CHECK(hipDeviceReset());
}

__constant__ int cK[1];
__constant__ int cMaxRowSize[1];
__constant__ int cRows[1];
__constant__ int cAlignFeat[2];
__constant__ float cRegLambda[1];
__constant__ float cLearningRate[1];
__constant__ int cWeightsOffset[1];
__constant__ int cBatchOffset[1];


template<typename T>
__global__ void wTxKernel(const int *__restrict__ features, const int *__restrict__ fields, const T *__restrict__ values,
                          const T *__restrict__ r, const int *__restrict__ rowSizes,
                          T *__restrict__ weightsPtr, T *__restrict__ losses, const bool update, const int *__restrict__ labels = nullptr) {
  int rowIdx = (blockIdx.x * blockDim.x + threadIdx.x) / cMaxRowSize[0];

  if(rowIdx >= cRows[0]) return;

  int rowSize = rowSizes[rowIdx + 1] - rowSizes[rowIdx];

  int nodeIdx = (blockIdx.x * blockDim.x + threadIdx.x) % cMaxRowSize[0];

  if(nodeIdx >= rowSize) return;

  __shared__ int fieldFeature[MAX_BLOCK_THREADS * 2];
  __shared__ T vals[MAX_BLOCK_THREADS];
  __shared__ T scales[MAX_BLOCK_THREADS];

  __shared__ T kappas[MAX_BLOCK_THREADS];
  __shared__ T expnyts[MAX_BLOCK_THREADS];

  int n1 = rowSizes[rowIdx] + nodeIdx - cBatchOffset[0];

  fieldFeature[threadIdx.x * 2] = fields[n1];
  fieldFeature[threadIdx.x * 2 + 1] = features[n1];
  vals[threadIdx.x] = values[n1];
  scales[rowIdx % MAX_BLOCK_THREADS] = r[rowIdx];

  if(update) {
    expnyts[rowIdx % MAX_BLOCK_THREADS] = std::exp(-labels[rowIdx] * losses[rowIdx]);
    kappas[rowIdx % MAX_BLOCK_THREADS] = -labels[rowIdx] * expnyts[rowIdx % MAX_BLOCK_THREADS] / (1 + expnyts[rowIdx % MAX_BLOCK_THREADS]);
  }

  __syncthreads();

  T loss = 0.0;

  for(int i = 1; n1 + i < rowSizes[rowIdx + 1] - cBatchOffset[0]; i++) {
    // We cache some of the field:feature:values in shared memory, only as many "nodes" as there are threads
    // so we know for 100% the initial node will be cached (since we run 1 thread per each starting node)
    // but the subsequent nodes can be within the same block or they can spill to consequitive blocks
    // depending on the size of the row and number of threads in a block
    const int idx1 = fieldFeature[threadIdx.x * 2 + 1] * cAlignFeat[0] +
        (threadIdx.x + i < MAX_BLOCK_THREADS ? fieldFeature[(threadIdx.x + i) * 2] : fields[n1 + i]) * cAlignFeat[1];

    const int idx2 = (threadIdx.x + i < MAX_BLOCK_THREADS ? fieldFeature[(threadIdx.x + i) * 2 + 1] : features[n1 + i]) * cAlignFeat[0] +
        fieldFeature[threadIdx.x * 2] * cAlignFeat[1];

    const T v = vals[threadIdx.x] * (threadIdx.x + i < MAX_BLOCK_THREADS ? vals[threadIdx.x + i] : values[n1 + i]) * scales[rowIdx % MAX_BLOCK_THREADS];

    if (update) {
      for (int d = 0; d < cK[0] * cWeightsOffset[0]; d+=cWeightsOffset[0]) {
        T w1d = (weightsPtr + idx1)[d];
        T w2d = (weightsPtr + idx2)[d];
        const T g1 = cRegLambda[0] * w1d + kappas[rowIdx % MAX_BLOCK_THREADS] * w2d * v;
        const T g2 = cRegLambda[0] * w2d + kappas[rowIdx % MAX_BLOCK_THREADS] * w1d * v;

        const T w1gdup = (weightsPtr + idx1)[d+1] + g1 * g1;
        const T w2gdup = (weightsPtr + idx2)[d+1] + g2 * g2;

        (weightsPtr + idx1)[d] -= cLearningRate[0] / std::sqrt(w1gdup) * g1;
        (weightsPtr + idx2)[d] -= cLearningRate[0] / std::sqrt(w2gdup) * g2;

        (weightsPtr + idx1)[d+1] = w1gdup;
        (weightsPtr + idx2)[d+1] = w2gdup;

      }
    } else {
      for (int d = 0; d < cK[0] * cWeightsOffset[0]; d+=cWeightsOffset[0]) {
        loss += (weightsPtr + idx1)[d] * (weightsPtr + idx2)[d] * v;
      }
    }
  }
  if(!update) {
    atomicAdd(losses + rowIdx, loss);
  }
}

template<typename T>
void Trainer<T>::predict(T *predictions) {
  for (int i = 0; i < params.nGpus; i++) {

    int initialBatchOffset = 0;
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(cBatchOffset), &initialBatchOffset, sizeof(int)));
    int record = 0;
    while (trainDataBatcher[i]->hasNext()) {
      DatasetBatch<T> *batch = trainDataBatcher[i]->nextBatch(this->params.batchSize);

      // TODO once per predictions and share
      T *losses;
      hipMalloc(&losses, batch->numRows * sizeof(T));

      int alignFeat1 = params.numFields * params.k;
      int alignFeat2 = params.k;

      int threads = MAX_BLOCK_THREADS;
      int maxRowSize = batch->widestRow();
      size_t totalThreads = batch->numRows * maxRowSize;
      int blocks = std::ceil((double)totalThreads / threads);

      T* weightsPtr = this->model->weightsRawPtr(i);

      CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(cK), &params.k, sizeof(int)));
      CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(cMaxRowSize), &maxRowSize, sizeof(int)));
      CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(cRows), &batch->numRows, sizeof(int)));
      int alignTmp[2] = { alignFeat1, alignFeat2 };
      CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(cAlignFeat), &alignTmp, 2 * sizeof(int)));
      CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(cRegLambda), &params.regLambda, sizeof(float)));
      CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(cLearningRate), &params.learningRate, sizeof(float)));
      int offset = 1;
      CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(cWeightsOffset), &offset, sizeof(int)));

      wTxKernel << < blocks, threads >> > (batch->features, batch->fields, batch->values, batch->scales, batch->rowPositions,
          weightsPtr, losses, false);

      CUDA_CHECK(hipMemcpy(predictions + record, losses, batch->numRows * sizeof(T), hipMemcpyDeviceToHost));

      CUDA_CHECK(hipDeviceSynchronize());
      CUDA_CHECK(hipGetLastError());

      record += batch->numRows;

      thrust::fill(thrust::device, losses, losses + batch->numRows, 0.0);

      CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(cBatchOffset), batch->rowPositions + batch->numRows, sizeof(int), 0, hipMemcpyDeviceToDevice));

      delete batch;
      hipFree(losses);
    }

    std::transform (predictions, predictions + params.numRows, predictions, [&](T val){ return 1.0 / (1.0 + std::exp(-val)); });
  }
}

template<typename T>
T Trainer<T>::oneEpoch(bool update) {
  Timer timer;
  log_debug(this->params.verbose, "Computing an FFM epoch (update = %s)", update ? "true" : "false");

  T loss = 0;

  for (int i = 0; i < params.nGpus; i++) {
    int initialBatchOffset = 0;
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(cBatchOffset), &initialBatchOffset, sizeof(int)));

    while (trainDataBatcher[i]->hasNext()) {
      /**
       * Get batch
       */

      timer.tic();
      DatasetBatch<T> *batch = trainDataBatcher[i]->nextBatch(this->params.batchSize);
      timer.toc();
      log_verbose(params.verbose, "Getting batch took %f.", timer.pop());

      // todo once per trainer and dealloc
      T *losses;
      hipMalloc(&losses, batch->numRows * sizeof(T));

      int alignFeat1 = params.numFields * params.k * 2;
      int alignFeat2 = params.k * 2;

      int threads = MAX_BLOCK_THREADS;
      int maxRowSize = batch->widestRow();
      size_t totalThreads = batch->numRows * maxRowSize;
      int blocks = std::ceil((double)totalThreads / threads);

      /**
        * Alloc
        */
      T* weightsPtr = this->model->weightsRawPtr(i);

      CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(cK), &params.k, sizeof(int)));
      CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(cMaxRowSize), &maxRowSize, sizeof(int)));
      CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(cRows), &batch->numRows, sizeof(int)));
      int alignTmp[2] = { alignFeat1, alignFeat2 };
      CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(cAlignFeat), &alignTmp, 2 * sizeof(int)));
      CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(cRegLambda), &params.regLambda, sizeof(float)));
      CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(cLearningRate), &params.learningRate, sizeof(float)));
      int offset = 2;
      CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(cWeightsOffset), &offset, sizeof(int)));

      timer.tic();

      wTxKernel << < blocks, threads>>> (batch->features, batch->fields, batch->values, batch->scales, batch->rowPositions,
              weightsPtr, losses, false);

      CUDA_CHECK(hipDeviceSynchronize());
      CUDA_CHECK(hipGetLastError());
      timer.toc();
      log_verbose(params.verbose, "wTx (false) took %f.", timer.pop());

      timer.tic();

      if (update) {
        wTxKernel << < blocks, threads>>> (batch->features, batch->fields, batch->values, batch->scales, batch->rowPositions,
                weightsPtr, losses, true, batch->labels);
      }

      CUDA_CHECK(hipDeviceSynchronize());
      CUDA_CHECK(hipGetLastError());
      timer.toc();
      log_verbose(params.verbose, "wTx (true) took %f.", timer.pop());

      timer.tic();
      int* labels = batch->labels;
      thrust::counting_iterator<int> counter(0);
      loss += thrust::transform_reduce(counter, counter + batch->numRows , [=]__device__(int idx) {
        return std::log(1 + std::exp(-labels[idx] * losses[idx]));
      },
      (T) 0.0, thrust::plus<T>());
      timer.toc();

      log_verbose(params.verbose, "Loss compute took %f.", timer.pop());

      thrust::fill(thrust::device, losses, losses + batch->numRows, 0.0);

      CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(cBatchOffset), batch->rowPositions + batch->numRows, sizeof(int), 0, hipMemcpyDeviceToDevice));

      delete batch;
      hipFree(losses);
    }

    trainDataBatcher[i]->reset();
  }

  if (params.nGpus != 1) {
    // TODO average local weights
  } // Don't do anything for 1GPU cases

  log_debug(this->params.verbose, "Log loss = %f", loss / params.numRows);

  return loss / params.numRows;
}

template<typename T>
bool Trainer<T>::earlyStop() {
  // TODO implement
  return false;
}

template
class Trainer<float>;
template
class Trainer<double>;

}