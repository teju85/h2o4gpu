#include "hip/hip_runtime.h"
/*!
 * Copyright 2018 H2O.ai, Inc.
 * License   Apache License Version 2.0 (see LICENSE for details)
 */
#include "../../base/ffm/trainer.h"
#include "batching_gpu.cuh"
#include <cmath>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <thrust/device_vector.h>

namespace ffm {

template<typename T>
Trainer<T>::Trainer(const Dataset<T> &dataset, Model<T> &model, Params const &params)
    : trainDataBatcher(1), model(model), params(params) {
  // TODO delete in destructor
  DatasetBatcherGPU<T> *batcher = new DatasetBatcherGPU<T>(dataset, params);
  trainDataBatcher[0] = batcher;
}

/**
 * Original ffm gradient/weight update method from https://github.com/guestwalk/libffm with slight adjustments
 */
template<typename T>
T wTx(Row<T> *row,
      thrust::device_vector<T> &weights,
      thrust::device_vector<T> &weightsGradient,
      Params params,
      T kappa = 0,
      bool update = false,
      int verbose = 0) {
  log_verbose(params.verbose, "KAPPA = %f", kappa);
  thrust::device_vector < Node<T> * > nodes(row->data);

  size_t alignFeat1 = params.numFields * params.k;
  size_t alignFeat2 = params.k;

  T loss = 0.0;

  T* weightsPtr = thrust::raw_pointer_cast(weights.data());
  T* weightsGradientPtr = thrust::raw_pointer_cast(weightsGradient.data());

  T r = params.normalize ? row->scale : 1.0;

//#pragma omp parallel for schedule(static) reduction(+: loss)
  for (size_t n1 = 0; n1 < row->size; n1++) {
    Node<T> *node1 = nodes[n1];

    loss += thrust::transform_reduce(nodes.begin() + n1 + 1, nodes.end(), [=]__device__(Node<T> * node2) {
      size_t feature1 = node1->featureIdx;
      size_t field1 = node1->fieldIdx;
      T value1 = node1->value;

      size_t feature2 = node2->featureIdx;
      size_t field2 = node2->fieldIdx;
      T value2 = node2->value;
      T localt = 0;

      if (feature1 >= params.numFeatures || field1 >= params.numFields ||
          feature2 >= params.numFeatures || field2 >= params.numFields) {
        return localt;
      }

      size_t idx1 = feature1 * alignFeat1 + field2 * alignFeat2;
      size_t idx2 = feature2 * alignFeat1 + field1 * alignFeat2;
      T *w1 = weightsPtr + idx1;
      T *w2 = weightsPtr + idx2;

      T v = value1 * value2 * r;

      if (update) {
        T *wg1 = weightsGradientPtr + idx1;
        T *wg2 = weightsGradientPtr + idx2;

        for (size_t d = 0; d < params.k; d++) {
          T g1 = params.regLambda * w1[d] + kappa * w2[d] * v;
          T g2 = params.regLambda * w2[d] + kappa * w1[d] * v;

          wg1[d] += g1 * g1;
          wg2[d] += g2 * g2;

          w1[d] -= params.learningRate / sqrt(wg1[d]) * g1;
          w2[d] -= params.learningRate / sqrt(wg2[d]) * g2;
        }
      } else {
        for (size_t d = 0; d < alignFeat2; d++) {
          localt += w1[d] * w2[d] * v;
        }
      }

      return localt;
    },
    (T) 0.0,
        thrust::plus<T>());
  }

  return loss;
}

template<typename T>
void Trainer<T>::predict(T *predictions) {
  for (int i = 0; i < params.nGpus; i++) {
    log_verbose(this->params.verbose, "Copying weights of size %zu to GPU %d for predictions", this->model.weights.size(), i);
    thrust::device_vector<T> localWeights(this->model.weights.begin(), this->model.weights.end());
    thrust::device_vector<T> localGradients(this->model.weights.size());

    int record = 0;
    while (trainDataBatcher[i]->hasNext()) {
      DatasetBatch<T> batch = trainDataBatcher[i]->nextBatch(this->params.batchSize);

      T loss = 0;
      // TODO parallelize somehow
      while (batch.hasNext()) {
        Row<T> *row = batch.nextRow();

        T t = wTx(row, localWeights, localGradients, this->params);

        predictions[record++] = 1 / (1 + exp(-t));
      }
    }
  }
}

template<typename T>
// TODO return loss
T Trainer<T>::oneEpoch(bool update) {
  log_debug(this->params.verbose, "Computing an FFM epoch (update = %s)", update ? "true" : "false");

  T loss = 0;

  std::vector<thrust::device_vector<T>> allLocalWeights(params.nGpus);
  std::vector<thrust::device_vector<T>> allLocalGradients(params.nGpus);
  for (int i = 0; i < params.nGpus; i++) {
    log_verbose(this->params.verbose, "Copying weights of size %zu to GPU %d", this->model.weights.size(), i);

    // TODO do only once for all iterations?
    allLocalWeights[i].resize(this->model.weights.size());
    thrust::copy(this->model.weights.begin(), this->model.weights.end(), allLocalWeights[i].begin() );
    allLocalGradients[i].resize(this->model.gradients.size());
    thrust::copy(this->model.gradients.begin(), this->model.gradients.end(), allLocalGradients[i].begin() );

    while (trainDataBatcher[i]->hasNext()) {
      DatasetBatch<T> batch = trainDataBatcher[i]->nextBatch(this->params.batchSize);

      // TODO parallelize somehow
      // TODO shuffle batch?
      while (batch.hasNext()) {
        Row<T> *row = batch.nextRow();

        T t = wTx(row, allLocalWeights[i], allLocalGradients[i], this->params);

        T expnyt = exp(-row->label * t);
        loss += log(1 + expnyt);

        if (update) {
          T kappa = -row->label * expnyt / (1 + expnyt);
          wTx(row, allLocalWeights[i], allLocalGradients[i], this->params, kappa, true);
        }
      }
    }
    trainDataBatcher[i]->reset();
  }

  if (params.nGpus != 1) {
    // TODO average local weights
    // TODO distribute gradients
  } else {
    thrust::copy(allLocalWeights[0].begin(), allLocalWeights[0].end(), this->model.weights.begin());
    thrust::copy(allLocalGradients[0].begin(), allLocalGradients[0].end(), this->model.gradients.begin());
  }

  log_debug(this->params.verbose, "Log loss = %f", loss / params.numRows);

  return loss / params.numRows;
}

template<typename T>
bool Trainer<T>::earlyStop() {
  // TODO implement
  return false;
}

template
class Trainer<float>;
template
class Trainer<double>;

}